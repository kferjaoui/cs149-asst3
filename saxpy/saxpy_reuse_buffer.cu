#include "hip/hip_runtime.h"
// saxpy.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

static float* g_dx = nullptr;
static float* g_dy = nullptr;
static float* g_dres = nullptr;
static int    g_allocN = 0;

__global__ void
saxpy_kernel(int N, float alpha,
             const float* __restrict__ x,
             const float* __restrict__ y,
             float* __restrict__ result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) result[index] = alpha * x[index] + y[index];
}

void saxpyCudaReuseDeviceBuffer(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate device buffers once and reuse
    if (N != g_allocN) {
        if (g_dx) { hipFree(g_dx); hipFree(g_dy); hipFree(g_dres); }
        hipMalloc(&g_dx,   N * sizeof(float));
        hipMalloc(&g_dy,   N * sizeof(float));
        hipMalloc(&g_dres, N * sizeof(float));
        g_allocN = N;
    }

    // Measure end-to-end (H2D + kernel + D2H) with events
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    hipMemcpy(g_dx, xarray, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_dy, yarray, N * sizeof(float), hipMemcpyHostToDevice);

    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, g_dx, g_dy, g_dres);

    hipMemcpy(resultarray, g_dres, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start); hipEventDestroy(stop);

    int totalBytes = sizeof(float) * 3 * N;
    auto GBPerSec = [](int bytes, float sec){
        return static_cast<float>(bytes) / (1024.f*1024.f*1024.f) / sec;
    };
    printf("End-to-end: %.3f ms\t[%.3f GB/s]\n",
           ms, GBPerSec(totalBytes, ms/1000.0f));

    // Error check
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: CUDA error: code=%d, %s\n",
                errCode, hipGetErrorString(errCode));
    }
}

// (Optional) add a cleanup you can call once at program end if desired:
void saxpyCleanup() {
    if (g_dx) { hipFree(g_dx); g_dx=nullptr; }
    if (g_dy) { hipFree(g_dy); g_dy=nullptr; }
    if (g_dres) { hipFree(g_dres); g_dres=nullptr; }
    g_allocN = 0;
}

void printCudaInfo() {

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
